// benchmark for block-sparse spmm

#include "block_sparse/spmm/spmm_op.h"
#include "block_sparse/spmm/spmm_library_decl.h"
#include "block_sparse/cuda_array.h"
#include "argparse_util.h"
#include "timing_util.h"


inline float benchmark(SpmmBlockwiseInitFn_t init_fn, 
    SpmmBlockwiseExecFn_t exec_fn,
    BlockwiseSpMatrix<half> &spmat, int N, half *B, half *D, 
    hipStream_t stream = NULL, int warmup=10, int repeat = 100)
{
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        std::cerr << "return due to previous error. ";
        return -1;
    }

    GpuTimer gpu_timer;
    
    SpmmBlockwiseOpState state = (*init_fn)(spmat, N, B, D);
    if (!state.initSuccess) {
        std::cerr << "return due to unsuccessful initialization. " << std::endl;
        return -1;
    }

    (*exec_fn)(state, stream);
    hipDeviceSynchronize();
    if (!isCudaSuccess(hipGetLastError())) {
        std::cerr << "kernel failed." << std::endl;
        return -1;
    }

    for (int i = 0; i < warmup + repeat; i++) {
        if (i == warmup) 
            gpu_timer.start();
        
        (*exec_fn)(state, stream);
    }
    gpu_timer.stop();


    if (!isCudaSuccess(hipGetLastError())) {
        std::cerr << "kernel failed." << std::endl;
        return -1;
    }
    float dur = gpu_timer.elapsed_msecs() / repeat;
    return dur;    
}

int main(int argc, const char** argv)
{
    int m, n, k;
    int pattern_code, block_sz;
    float density;
    unsigned seed; 
    bool row_permute;
    bool load_pattern, store_pattern;
    std::string load_path, store_path;
    parseSpmmArgs(argc, argv, m, n, k, density, seed, pattern_code,
    block_sz, row_permute, store_pattern, store_path, load_pattern, load_path,
    /* verbose */true);
    
    // pad shapes
    const int n_pad_to = 32;
    if (m % block_sz != 0) {
        if (load_pattern) {
            std::cerr << "Loaded matrix shape nrow is not padded to block_sz\n";
            exit(EXIT_FAILURE);
        }
        else {
            m += (block_sz - (m % block_sz));
            std::cerr << "m padded to : " << m << "\n";
        }
    }
    if (n % n_pad_to != 0) {
        n += (n_pad_to - (n % n_pad_to));
        std::cerr << "n padded to : " << n << " for better alignment.\n";
    }

    CudaRandomArray<half> B;
    CudaZerosArray<half> D;
    B.initialize(k*n);
    D.initialize(m*n);
    B.sync_device();
    D.sync_device();
    
    // branch on pattern_code
    if (pattern_code == 0) { // blockwise

        BlockwiseSpMatrix<half> spmat;
        // if (load_pattern) {
        //     spmat.load_blockwise_matrix(load_path.c_str());
        // }
        // else {
            spmat.init_random(m, k, block_sz, 1, density, row_permute, seed);
        // }
        // if (store_pattern) {
        //     spmat.store_blockwise_matrix(store_path.c_str());
        // }

        spmat.transform_and_sync_device();

        // benchmark
        float gflop_count = (float)m / 1e9 * n*k*2;

#define BENCHMARK(BLOCK_SZ, BN, BK, WM, WN, WK, MMA_M, MMA_N, MMA_K, NSTAGE) \
{\
    std::cout << GPU_CC << " " << spmat.config_str << " ";\
    printf("%d %d %d %d %d %d %d %d %d %d %d ", n, BLOCK_SZ, BN, BK, WM, WN, WK, MMA_M, MMA_N, MMA_K, NSTAGE);\
    float dur = benchmark( \
            NAME_FUN(SpmmBlockwise, Init, BLOCK_SZ, BN, BK, WM, WN, WK, MMA_M, MMA_N, MMA_K, NSTAGE), \
            NAME_FUN(SpmmBlockwise, Exec, BLOCK_SZ, BN, BK, WM, WN, WK, MMA_M, MMA_N, MMA_K, NSTAGE), \
            spmat, n, B.device_ptr, D.device_ptr); \
    printf("%f (us) %f (effective tflop/s)\n", dur*1e3, gflop_count/dur); \
}

        #if GPU_CC >= 80
        switch (block_sz) {
            case 16:
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 16, 2);
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 16, 3);
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 16, 4);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 16, 2);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 16, 3);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 16, 4);
            BENCHMARK(16, 128,16, 16, 16, 16, 16, 16, 16, 2);
            BENCHMARK(16, 128,16, 16, 16, 16, 16, 16, 16, 3);
            BENCHMARK(16, 128,16, 16, 32, 16, 16, 16, 16, 4);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 16, 2);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 16, 3);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 16, 4);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 16, 2);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 16, 3);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 16, 2);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 16, 3);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 16, 4);
            break;
            
            case 32:
            BENCHMARK(32, 64, 16, 32, 16, 16, 16, 16, 16, 2);
            BENCHMARK(32, 64, 16, 32, 16, 16, 16, 16, 16, 3);
            BENCHMARK(32, 64, 16, 32, 16, 16, 16, 16, 16, 4);
            BENCHMARK(32, 64, 32, 32, 16, 32, 16, 16, 16, 2);
            BENCHMARK(32, 64, 32, 32, 16, 32, 16, 16, 16, 3);
            BENCHMARK(32, 64, 32, 32, 16, 32, 16, 16, 16, 4);
            BENCHMARK(32, 128,16, 32, 32, 16, 16, 16, 16, 2);
            BENCHMARK(32, 128,16, 32, 32, 16, 16, 16, 16, 3);
            BENCHMARK(32, 128,16, 32, 32, 16, 16, 16, 16, 4);
            BENCHMARK(32, 128,32, 32, 32, 32, 16, 16, 16, 2);
            BENCHMARK(32, 128,32, 32, 32, 32, 16, 16, 16, 3);
            BENCHMARK(32, 128,32, 32, 32, 32, 16, 16, 16, 4);
            BENCHMARK(32, 128,64, 32, 32, 64, 16, 16, 16, 2);
            BENCHMARK(32, 256,16, 32, 64, 16, 16, 16, 16, 2);
            BENCHMARK(32, 256,16, 32, 64, 16, 16, 16, 16, 3);
            BENCHMARK(32, 256,32, 32, 64, 32, 16, 16, 16, 2);
            BENCHMARK(32, 256,32, 32, 64, 32, 16, 16, 16, 3);
            BENCHMARK(32, 256,32, 32, 64, 32, 16, 16, 16, 4);
            break;

            case 64:
            BENCHMARK(64, 64, 16, 32, 32, 16, 16, 16, 16, 2);
            BENCHMARK(64, 64, 16, 32, 32, 16, 16, 16, 16, 3);
            BENCHMARK(64, 64, 16, 32, 32, 16, 16, 16, 16, 4);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 16, 2);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 16, 3);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 16, 4);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 16, 5);
            BENCHMARK(64, 128,32, 32, 32, 32, 16, 16, 16, 2);
            BENCHMARK(64, 128,32, 32, 32, 32, 16, 16, 16, 3);
            BENCHMARK(64, 128,32, 32, 32, 32, 16, 16, 16, 4);
            BENCHMARK(64, 128,32, 64, 32, 32, 16, 16, 16, 2);
            BENCHMARK(64, 128,32, 64, 32, 32, 16, 16, 16, 3);
            BENCHMARK(64, 128,32, 64, 32, 32, 16, 16, 16, 4);
            BENCHMARK(64, 256,16, 32, 64, 16, 16, 16, 16, 2);
            BENCHMARK(64, 256,16, 32, 64, 16, 16, 16, 16, 3);
            break;
            
            case 128:
            BENCHMARK(128,32, 32, 32, 32, 32, 16, 16, 16, 2);
            BENCHMARK(128,32, 32, 32, 32, 32, 16, 16, 16, 3);
            BENCHMARK(128,64, 32, 16, 64, 32, 16, 16, 16, 2);
            BENCHMARK(128,64, 32, 16, 64, 32, 16, 16, 16, 3);
            BENCHMARK(128,64, 32, 16, 64, 32, 16, 16, 16, 4);
            BENCHMARK(128,64, 32, 32, 32, 32, 16, 16, 16, 2);
            BENCHMARK(128,64, 32, 32, 32, 32, 16, 16, 16, 3);
            BENCHMARK(128,64, 32, 32, 32, 32, 16, 16, 16, 4);
            BENCHMARK(128,64, 32, 32, 64, 32, 16, 16, 16, 2);
            BENCHMARK(128,64, 32, 32, 64, 32, 16, 16, 16, 3);
            BENCHMARK(128,64, 32, 32, 64, 32, 16, 16, 16, 4);
            BENCHMARK(128,128,32, 32, 64, 32, 16, 16, 16, 2);
            BENCHMARK(128,128,32, 32, 64, 32, 16, 16, 16, 4);
            BENCHMARK(128,128,32, 64, 32, 32, 16, 16, 16, 2);
            BENCHMARK(128,128,32, 64, 32, 32, 16, 16, 16, 4);        
            break;
        }
        #else 
        #if GPU_CC >= 75
        switch (block_sz) {
            case 16:
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 8, 2);
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 8, 3);
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 8, 4);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 8, 2);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 8, 3);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 8, 4);
            BENCHMARK(16, 128,16, 16, 16, 16, 16, 16, 8, 2);
            BENCHMARK(16, 128,16, 16, 16, 16, 16, 16, 8, 3);
            BENCHMARK(16, 128,16, 16, 32, 16, 16, 16, 8, 4);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 8, 2);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 8, 3);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 8, 4);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 8, 2);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 8, 3);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 8, 2);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 8, 3);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 8, 4);
            break; case 32:
            BENCHMARK(32, 64, 16, 32, 16, 16, 16, 16, 8, 2);
            BENCHMARK(32, 64, 16, 32, 16, 16, 16, 16, 8, 3);
            BENCHMARK(32, 64, 16, 32, 16, 16, 16, 16, 8, 4);
            BENCHMARK(32, 64, 32, 32, 16, 32, 16, 16, 8, 2);
            BENCHMARK(32, 64, 32, 32, 16, 32, 16, 16, 8, 3);
            BENCHMARK(32, 64, 32, 32, 16, 32, 16, 16, 8, 4);
            BENCHMARK(32, 128,16, 32, 32, 16, 16, 16, 8, 2);
            BENCHMARK(32, 128,16, 32, 32, 16, 16, 16, 8, 3);
            BENCHMARK(32, 128,16, 32, 32, 16, 16, 16, 8, 4);
            BENCHMARK(32, 128,32, 32, 32, 32, 16, 16, 8, 2);
            BENCHMARK(32, 128,32, 32, 32, 32, 16, 16, 8, 3);
            BENCHMARK(32, 128,32, 32, 32, 32, 16, 16, 8, 4);
            BENCHMARK(32, 128,64, 32, 32, 64, 16, 16, 8, 2);
            BENCHMARK(32, 256,16, 32, 64, 16, 16, 16, 8, 2);
            BENCHMARK(32, 256,16, 32, 64, 16, 16, 16, 8, 3);
            BENCHMARK(32, 256,32, 32, 64, 32, 16, 16, 8, 2);
            BENCHMARK(32, 256,32, 32, 64, 32, 16, 16, 8, 3);
            BENCHMARK(32, 256,32, 32, 64, 32, 16, 16, 8, 4);
            break; case 64:
            BENCHMARK(64, 64, 16, 32, 32, 16, 16, 16, 8, 2);
            BENCHMARK(64, 64, 16, 32, 32, 16, 16, 16, 8, 3);
            BENCHMARK(64, 64, 16, 32, 32, 16, 16, 16, 8, 4);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 8, 2);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 8, 3);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 8, 4);
            BENCHMARK(64, 64, 32, 32, 32, 32, 16, 16, 8, 5);
            BENCHMARK(64, 128,32, 32, 32, 32, 16, 16, 8, 2);
            BENCHMARK(64, 128,32, 32, 32, 32, 16, 16, 8, 3);
            BENCHMARK(64, 128,32, 32, 32, 32, 16, 16, 8, 4);
            BENCHMARK(64, 128,32, 64, 32, 32, 16, 16, 8, 2);
            BENCHMARK(64, 128,32, 64, 32, 32, 16, 16, 8, 3);
            BENCHMARK(64, 128,32, 64, 32, 32, 16, 16, 8, 4);
            BENCHMARK(64, 256,16, 32, 64, 16, 16, 16, 8, 2);
            BENCHMARK(64, 256,16, 32, 64, 16, 16, 16, 8, 3);
            break; case 128:
            BENCHMARK(128,32, 32, 32, 32, 32, 16, 16, 8, 2);
            BENCHMARK(128,32, 32, 32, 32, 32, 16, 16, 8, 3);
            BENCHMARK(128,64, 32, 16, 64, 32, 16, 16, 8, 2);
            BENCHMARK(128,64, 32, 16, 64, 32, 16, 16, 8, 3);
            BENCHMARK(128,64, 32, 16, 64, 32, 16, 16, 8, 4);
            BENCHMARK(128,64, 32, 32, 32, 32, 16, 16, 8, 2);
            BENCHMARK(128,64, 32, 32, 32, 32, 16, 16, 8, 3);
            BENCHMARK(128,64, 32, 32, 32, 32, 16, 16, 8, 4);
            BENCHMARK(128,64, 32, 32, 64, 32, 16, 16, 8, 2);
            BENCHMARK(128,64, 32, 32, 64, 32, 16, 16, 8, 3);
            BENCHMARK(128,64, 32, 32, 64, 32, 16, 16, 8, 4);
            BENCHMARK(128,128,32, 32, 64, 32, 16, 16, 8, 2);
            BENCHMARK(128,128,32, 32, 64, 32, 16, 16, 8, 4);
            BENCHMARK(128,128,32, 64, 32, 32, 16, 16, 8, 2);
            BENCHMARK(128,128,32, 64, 32, 32, 16, 16, 8, 4);
        }
        #else
        switch(block_sz) {
            case 16:
            BENCHMARK(16, 64, 16, 16, 16, 16, 16, 16, 16, 2);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 16, 2);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 16, 3);
            BENCHMARK(16, 64, 32, 16, 16, 32, 16, 16, 16, 4);
            BENCHMARK(16, 128,16, 16, 32, 16, 16, 16, 16, 2);
            BENCHMARK(16, 128,16, 16, 32, 16, 16, 16, 16, 3);
            BENCHMARK(16, 128,16, 16, 32, 16, 16, 16, 16, 4);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 16, 2);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 16, 3);
            BENCHMARK(16, 128,32, 16, 32, 32, 16, 16, 16, 4);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 16, 2);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 16, 3);
            BENCHMARK(16, 256,16, 16, 64, 16, 16, 16, 16, 4);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 16, 2);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 16, 3);
            BENCHMARK(16, 256,32, 16, 64, 32, 16, 16, 16, 4);
            break; case 32:
            BENCHMARK(32, 128,16, 32, 32, 16, 32, 32, 16, 2);
            BENCHMARK(32, 128,16, 32, 32, 16, 32, 32, 16, 3);
            BENCHMARK(32, 128,16, 32, 32, 16, 32, 32, 16, 4);
            BENCHMARK(32, 128,32, 16, 64, 32, 16, 16, 16, 2);
            BENCHMARK(32, 128,32, 16, 64, 32, 16, 16, 16, 3);
            BENCHMARK(32, 128,32, 16, 64, 32, 16, 16, 16, 4);
            BENCHMARK(32, 128,32, 32, 32, 32, 32, 32, 16, 2);
            BENCHMARK(32, 128,32, 32, 32, 32, 32, 32, 16, 3);
            BENCHMARK(32, 128,32, 32, 32, 32, 32, 32, 16, 4);
            BENCHMARK(32, 256,16, 32, 64, 16, 32, 32, 16, 2);
            BENCHMARK(32, 256,16, 32, 64, 16, 32, 32, 16, 3);
            BENCHMARK(32, 256,16, 32, 64, 16, 32, 32, 16, 4);
            BENCHMARK(32, 256,32, 32, 64, 32, 32, 32, 16, 2);
            BENCHMARK(32, 256,32, 32, 64, 32, 32, 32, 16, 3);
            BENCHMARK(32, 256,32, 32, 64, 32, 32, 32, 16, 4);
            break; case 64:
            BENCHMARK(64, 64, 16, 32, 32, 16, 32, 32, 16, 2);
            BENCHMARK(64, 64, 16, 32, 32, 16, 32, 32, 16, 3);
            BENCHMARK(64, 64, 16, 32, 32, 16, 32, 32, 16, 4);
            BENCHMARK(64, 64, 32, 32, 32, 32, 32, 32, 16, 2);
            BENCHMARK(64, 64, 32, 32, 32, 32, 32, 32, 16, 3);
            BENCHMARK(64, 64, 32, 32, 32, 32, 32, 32, 16, 4);
            BENCHMARK(64, 128,16, 32, 64, 16, 32, 32, 16, 2);
            BENCHMARK(64, 128,16, 32, 64, 16, 32, 32, 16, 3);
            BENCHMARK(64, 128,16, 32, 64, 16, 32, 32, 16, 4);
            BENCHMARK(64, 128,32, 32, 64, 32, 32, 32, 16, 2);
            BENCHMARK(64, 128,32, 32, 64, 32, 32, 32, 16, 3);
            BENCHMARK(64, 128,32, 32, 64, 32, 32, 32, 16, 4);
            BENCHMARK(64, 128,16, 64, 32, 16, 32, 32, 16, 2);
            BENCHMARK(64, 128,16, 64, 32, 16, 32, 32, 16, 3);
            BENCHMARK(64, 128,16, 64, 32, 16, 32, 32, 16, 4);
            BENCHMARK(64, 128,32, 64, 32, 32, 32, 32, 16, 2);
            BENCHMARK(64, 128,32, 64, 32, 32, 32, 32, 16, 3);
            BENCHMARK(64, 128,32, 64, 32, 32, 32, 32, 16, 4);
            break; case 128:
            BENCHMARK(128,32, 32, 32, 32, 32, 32, 32, 16, 2);
            BENCHMARK(128,32, 32, 32, 32, 32, 32, 32, 16, 3);
            BENCHMARK(128,64, 32, 32, 64, 32, 32, 32, 16, 2);
            BENCHMARK(128,64, 32, 32, 64, 32, 32, 32, 16, 3);
            BENCHMARK(128,64, 32, 32, 64, 32, 32, 32, 16, 4);
            BENCHMARK(128,128,32, 32, 64, 32, 32, 32, 16, 2);
            BENCHMARK(128,128,32, 32, 64, 32, 32, 32, 16, 4);
            BENCHMARK(128,128,32, 64, 32, 32, 32, 32, 16, 2);
            BENCHMARK(128,128,32, 64, 32, 32, 32, 32, 16, 4);
            BENCHMARK(128,128,64, 32, 64, 64, 32, 32, 16, 2);
            BENCHMARK(128,128,64, 64, 32, 64, 32, 32, 16, 2);        
        }
        #endif  // GPU_CC >= 75
        #endif  // GPU_CC >= 80
    }
    else { // block-2in4
        std::cerr << "only pattern:block is implemented.\n";
        exit(EXIT_FAILURE);
    }
}